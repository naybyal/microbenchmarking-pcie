// benchmark.cu
// A simple PCIe Bandwidth Benchmark using CUDA runtime API.
//
// This program allocates a 64 MB pinned host buffer and a device buffer,
// then performs multiple iterations of data transfers in both directions.
// It uses CUDA events to time the transfers and calculates the average bandwidth.
//
// Usage:
//   ./cuda_pcie_bw [iterations] [buffer_size_in_MB]
//   Default iterations: 10, default buffer size: 64 MB

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define DEFAULT_ITERATIONS 10
#define DEFAULT_BUF_SIZE_MB 64

// Macro for checking CUDA errors.
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(int argc, char* argv[]) {
    int iterations = DEFAULT_ITERATIONS;
    size_t bufSize = DEFAULT_BUF_SIZE_MB * 1024 * 1024; // 64 MB

    if(argc >= 2) {
        iterations = atoi(argv[1]);
        if(iterations <= 0) iterations = DEFAULT_ITERATIONS;
    }
    if(argc >= 3) {
        bufSize = atol(argv[2]) * 1024 * 1024;
        if(bufSize == 0) bufSize = DEFAULT_BUF_SIZE_MB * 1024 * 1024;
    }

    printf("CUDA PCIe Bandwidth Benchmark\n");
    printf("Buffer Size: %zu bytes (%zu MB), Iterations: %d\n\n",
           bufSize, bufSize / (1024 * 1024), iterations);
    cudaCheckError( hipSetDevice(0) );
    // Allocate pinned host memory for optimal transfer performance.
    void* hostData;
    cudaCheckError( hipHostMalloc(&hostData, bufSize, hipHostMallocDefault) );
    memset(hostData, 0xA5, bufSize);

    // Allocate device memory.
    void* deviceData;
    cudaCheckError( hipMalloc(&deviceData, bufSize) );

    // Create CUDA events for timing.
    hipEvent_t start, stop;
    cudaCheckError( hipEventCreate(&start) );
    cudaCheckError( hipEventCreate(&stop) );

    float totalH2D_ms = 0.0f, totalD2H_ms = 0.0f;
    float elapsed_ms = 0.0f;

    // Benchmark Host-to-Device transfers.
    for (int i = 0; i < iterations; i++) {
        cudaCheckError( hipEventRecord(start, 0) );
        cudaCheckError( hipMemcpy(deviceData, hostData, bufSize, hipMemcpyHostToDevice) );
        cudaCheckError( hipEventRecord(stop, 0) );
        cudaCheckError( hipEventSynchronize(stop) );
        cudaCheckError( hipEventElapsedTime(&elapsed_ms, start, stop) );
        totalH2D_ms += elapsed_ms;
    }

    // Benchmark Device-to-Host transfers.
    for (int i = 0; i < iterations; i++) {
        cudaCheckError( hipEventRecord(start, 0) );
        cudaCheckError( hipMemcpy(hostData, deviceData, bufSize, hipMemcpyDeviceToHost) );
        cudaCheckError( hipEventRecord(stop, 0) );
        cudaCheckError( hipEventSynchronize(stop) );
        cudaCheckError( hipEventElapsedTime(&elapsed_ms, start, stop) );
        totalD2H_ms += elapsed_ms;
    }

    // Compute average times and bandwidths.
    float avgH2D_sec = (totalH2D_ms / iterations) / 1000.0f;
    float avgD2H_sec = (totalD2H_ms / iterations) / 1000.0f;
    // Convert bytes to gigabytes.
    double gbTransferred = (double)bufSize / (1LL << 30);
    double h2dBandwidth = gbTransferred / avgH2D_sec;
    double d2hBandwidth = gbTransferred / avgD2H_sec;

    printf("Average Host->Device Bandwidth: %.2f GB/s\n", h2dBandwidth);
    printf("Average Device->Host Bandwidth: %.2f GB/s\n", d2hBandwidth);

    // Cleanup resources.
    cudaCheckError( hipEventDestroy(start) );
    cudaCheckError( hipEventDestroy(stop) );
    cudaCheckError( hipFree(deviceData) );
    cudaCheckError( hipHostFree(hostData) );

    return 0;
}

